#include "hip/hip_runtime.h"
/**********************************************************************
 *  test_1d_compare_cufft.cu
 *  -------------------------------------------------------------------
 *  Benchmarks:
 *      • fft_cpu        – iterative radix-2 FFT on host    (single-prec.)
 *      • fft_gpu        – your custom CUDA implementation (fft.cu)
 *      • cuFFT          – NVIDIA’s library implementation (hipfftExecC2C)
 *
 *  Prints run times in milliseconds for input sizes N = 2^k.
 *  Timing strategy
 *      – CPU code:          std::chrono::high_resolution_clock
 *      – GPU code:          hipEvent_t (includes H↔D copies + exec)
 *********************************************************************/

#include <chrono>
#include <iomanip>
#include <iostream>
#include <limits>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft.cu"                    // <- your routines

/* ------------ simple CUDA / cuFFT error helpers ------------------ */
#define CHECK_CUDA(call)                                             \
    do {                                                             \
        hipError_t _e = (call);                                     \
        if (_e != hipSuccess) {                                     \
            std::cerr << "CUDA error " << hipGetErrorString(_e)     \
                      << " at " << __FILE__ << ':' << __LINE__       \
                      << std::endl;                                  \
            std::exit(EXIT_FAILURE);                                 \
        }                                                            \
    } while (0)

#define CHECK_CUFFT(call)                                            \
    do {                                                             \
        hipfftResult _e = (call);                                     \
        if (_e != HIPFFT_SUCCESS) {                                   \
            std::cerr << "cuFFT error " << _e                        \
                      << " at " << __FILE__ << ':' << __LINE__       \
                      << std::endl;                                  \
            std::exit(EXIT_FAILURE);                                 \
        }                                                            \
    } while (0)

int main()
{
    using clock   = std::chrono::high_resolution_clock;
    using dur_ms  = std::chrono::duration<double, std::milli>;

    std::cout << std::left
              << std::setw(12) << "N"
              << std::setw(15) << "FFT_CPU(ms)"
              << std::setw(15) << "FFT_GPU(ms)"
              << std::setw(15) << "cuFFT(ms)"     << '\n';

    /* ------------------------------------------------------------------
       loop over power-of-two sizes
    ------------------------------------------------------------------ */
    for (int exp = 3; exp <= 20; ++exp)
    {
        std::size_t N = 1ULL << exp;
        std::size_t bytes = N * sizeof(hipFloatComplex);

        /* ---- host-side buffers ------------------------------------ */
        std::vector<std::complex<float>> x(N),
                                         y_cpu(N),     // fft_cpu result
                                         y_gpu(N),     // custom CUDA
                                         y_cufft(N);   // cuFFT

        /* test signal (non-trivial, deterministic) ------------------ */
        for (std::size_t i = 0; i < N; ++i)
            x[i] = { std::sin(0.017453292f * static_cast<float>(i)),
                     std::cos(0.031415927f * static_cast<float>(i)) };

        /* -------------- CPU FFT timing ----------------------------- */
        auto  t0_cpu = clock::now();
        fft_cpu(x.data(), y_cpu.data(), N);
        auto  t1_cpu = clock::now();
        double ms_cpu = dur_ms(t1_cpu - t0_cpu).count();

        /* -----------------------------------------------------------
           GPU 1: custom fft_gpu   (wrapper allocates + copies)
        ----------------------------------------------------------- */
        auto  t0_gpu = clock::now();
        fft_gpu(x.data(), y_gpu.data(), N);
        CHECK_CUDA(hipDeviceSynchronize());
        auto  t1_gpu = clock::now();
        double ms_gpu = dur_ms(t1_gpu - t0_gpu).count();

        /* -----------------------------------------------------------
           GPU 2: cuFFT – we time: H→D copy + exec + D→H copy
        ----------------------------------------------------------- */
        hipFloatComplex *d_data = nullptr;
        CHECK_CUDA(hipMalloc(&d_data, bytes));

        hipEvent_t evStart, evStop;
        CHECK_CUDA(hipEventCreate(&evStart));
        CHECK_CUDA(hipEventCreate(&evStop));

        /* host → device */
        CHECK_CUDA(hipMemcpy(d_data, x.data(), bytes,
                              hipMemcpyHostToDevice));

        /* plan – its creation time is *not* included in timing
           (cuFFT documentation recommends re-using plans)           */
        hipfftHandle plan;
        CHECK_CUFFT(hipfftPlan1d(&plan,
                                static_cast<int>(N),
                                HIPFFT_C2C,
                                /*batch*/ 1));

        CHECK_CUDA(hipEventRecord(evStart));
        CHECK_CUFFT(hipfftExecC2C(plan,
                                 d_data, d_data,
                                 HIPFFT_FORWARD));        // sign = –1
        CHECK_CUDA(hipEventRecord(evStop));
        CHECK_CUDA(hipEventSynchronize(evStop));

        /* device → host */
        CHECK_CUDA(hipMemcpy(y_cufft.data(), d_data, bytes,
                              hipMemcpyDeviceToHost));

        float ms_cufft = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&ms_cufft, evStart, evStop));

        /* cleanup */
        hipfftDestroy(plan);
        hipFree(d_data);
        hipEventDestroy(evStart);
        hipEventDestroy(evStop);

        /* -------------- print line --------------------------------- */
        std::cout << std::setw(12) << N
                  << std::setw(15) << std::fixed << std::setprecision(3) << ms_cpu
                  << std::setw(15) << std::fixed << std::setprecision(3) << ms_gpu
                  << std::setw(15) << std::fixed << std::setprecision(3) << ms_cufft
                  << '\n';
    }
    return 0;
}
