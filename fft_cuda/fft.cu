﻿/* ================================================================== *
   Fast Fourier Transform (single-precision, radix-2, CUDA & CPU)
   — unified forward / inverse kernels (sign = ±1)
 * ================================================================== */

 #include <hip/hip_runtime.h>
 #include <hip/hip_complex.h>
 
 #define _USE_MATH_DEFINES
 #include <cmath>
 #include <complex>
 #include <iostream>
 #include <vector>
 
 #ifndef M_PI
 #define M_PI 3.14159265358979323846f
 #endif
 
 
 /* ------------------------------------------------------------------ *
    Bit-reversal: used by both host & device code
  * ------------------------------------------------------------------ */
 __host__ __device__
 unsigned bit_reverse(unsigned v, int lgN)
 {
     v = ((v & 0x55555555u) << 1) | ((v & 0xAAAAAAAAu) >> 1);
     v = ((v & 0x33333333u) << 2) | ((v & 0xCCCCCCCCu) >> 2);
     v = ((v & 0x0F0F0F0Fu) << 4) | ((v & 0xF0F0F0F0u) >> 4);
     v = ((v & 0x00FF00FFu) << 8) | ((v & 0xFF00FF00u) >> 8);
     v = (v << 16) | (v >> 16);
     return v >> (32 - lgN);
 }
 
 
 /* ================================================================== *
                              CPU  ROUTINES
  * ================================================================== */
 
 /* ---------- simple, slow DFT (unchanged) -------------------------- */
 int dft_cpu(const std::complex<float>* in, std::complex<float>* out,
             std::size_t N)
 {
     for (std::size_t k = 0; k < N; ++k) {
         std::complex<float> sum = 0.0f;
         for (std::size_t n = 0; n < N; ++n)
             sum += in[n] *
                    std::exp(std::complex<float>(0.0f,
                           -2.f * M_PI * static_cast<float>(k * n) / N));
         out[k] = sum;
     }
     return 0;
 }
 
 /* ---------- generic iterative FFT (dir = −1 fwd, +1 inv) --------- */
 static int fft_cpu_generic(const std::complex<float>* input,
                            std::complex<float>*       output,
                            std::size_t                N,
                            float                      dir)
 {
     if (N == 0 || (N & (N - 1))) return -1;      /* must be power of two */
 
     int lgN = static_cast<int>(std::log2(N));
 
     for (std::size_t i = 0; i < N; ++i)
         output[bit_reverse(i, lgN)] = input[i];
 
     for (int m = 2; m <= static_cast<int>(N); m <<= 1) {
         std::size_t mh = m >> 1;
         std::complex<float> w_m =
             std::exp(std::complex<float>(0.0f, dir * M_PI)
                      / static_cast<float>(mh));
 
         for (std::size_t k = 0; k < N; k += m) {
             std::complex<float> w = 1.0f;
             for (std::size_t j = 0; j < mh; ++j) {
                 auto u = output[k + j];
                 auto t = w * output[k + j + mh];
                 output[k + j]       = u + t;
                 output[k + j + mh]  = u - t;
                 w *= w_m;
             }
         }
     }
     return 0;
 }
 
 /* ---------- public wrappers -------------------------------------- */
 int  fft_cpu (const std::complex<float>* in,
               std::complex<float>*       out,
               std::size_t                N)
 { return fft_cpu_generic(in, out, N, -1.f); }
 
 int  ifft_cpu(const std::complex<float>* in,
               std::complex<float>*       out,
               std::size_t                N)
 {
     int err = fft_cpu_generic(in, out, N, +1.f);
     if (err) return err;
     float s = 1.f / static_cast<float>(N);
     for (std::size_t i = 0; i < N; ++i) out[i] *= s;
     return 0;
 }
 
 /* ---------- 2-D helpers ------------------------------------------ */
 static int fft2d_cpu_generic(const std::complex<float>* in,
                              std::complex<float>*       out,
                              std::size_t                rows,
                              std::size_t                cols,
                              float                      dir)
 {
     if (rows == 0 || cols == 0
         || (rows & (rows - 1)) || (cols & (cols - 1)))
         return -1;
 
     std::vector<std::complex<float>> tmp(rows * cols);
 
     /* row transforms */
     for (std::size_t r = 0; r < rows; ++r)
         fft_cpu_generic(in + r * cols, tmp.data() + r * cols, cols, dir);
 
     /* column transforms */
     std::vector<std::complex<float>> col_in(rows), col_out(rows);
     for (std::size_t c = 0; c < cols; ++c) {
         for (std::size_t r = 0; r < rows; ++r)
             col_in[r] = tmp[r * cols + c];
 
         fft_cpu_generic(col_in.data(), col_out.data(), rows, dir);
 
         for (std::size_t r = 0; r < rows; ++r)
             out[r * cols + c] = col_out[r];
     }
     return 0;
 }
 
 int  fft2d_cpu (const std::complex<float>* in,
                 std::complex<float>*       out,
                 std::size_t                rows,
                 std::size_t                cols)
 { return fft2d_cpu_generic(in, out, rows, cols, -1.f); }
 
 int  ifft2d_cpu(const std::complex<float>* in,
                 std::complex<float>*       out,
                 std::size_t                rows,
                 std::size_t                cols)
 {
     int err = fft2d_cpu_generic(in, out, rows, cols, +1.f);
     if (err) return err;
     float s = 1.f / static_cast<float>(rows * cols);
     for (std::size_t i = 0, n = rows * cols; i < n; ++i) out[i] *= s;
     return 0;
 }
 
 
 /* ================================================================== *
                            GPU   K E R N E L S
  * ================================================================== */
 
 /* ---------- unified radix-2 butterfly (dir = ±1) ------------------ */
 __global__ void fft_stage_kernel(hipFloatComplex* data,
                                  std::size_t     N,
                                  int             stage,
                                  float           dir)
 {
     unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
     if (tid >= (N >> 1)) return;              /* only N/2 butterflies */
 
     std::size_t mh = 1u << (stage - 1);
     std::size_t m  = mh << 1;
 
     std::size_t k = (tid / mh) * m;
     std::size_t j = tid % mh;
 
     float angle = dir * M_PI * static_cast<float>(j) / static_cast<float>(mh);
     float sr, cr;  sincosf(angle, &sr, &cr);
     hipFloatComplex w = make_hipFloatComplex(cr, sr);
 
     hipFloatComplex u = data[k + j];
     hipFloatComplex t = hipCmulf(w, data[k + j + mh]);
     data[k + j]      = hipCaddf(u, t);
     data[k + j + mh] = hipCsubf(u, t);
 }
 
 /* ---------- row-wise butterfly ----------------------------------- */
 __global__ void row_fft_stage_kernel(hipFloatComplex* data,
                                      std::size_t     rows,
                                      std::size_t     cols,
                                      int             stage,
                                      float           dir)
 {
     std::size_t halfPerRow = cols >> 1;
     std::size_t tid  = blockIdx.x * blockDim.x + threadIdx.x;
     std::size_t work = rows * halfPerRow;
     if (tid >= work) return;
 
     std::size_t row = tid / halfPerRow;
     std::size_t off = tid % halfPerRow;
 
     std::size_t mh = 1u << (stage - 1);
     std::size_t m  = mh << 1;
 
     std::size_t k = (off / mh) * m;
     std::size_t j =  off % mh;
 
     float angle = dir * M_PI * static_cast<float>(j) / static_cast<float>(mh);
     float sr, cr;  sincosf(angle, &sr, &cr);
     hipFloatComplex w = make_hipFloatComplex(cr, sr);
 
     std::size_t base = row * cols;
     hipFloatComplex u = data[base + k + j];
     hipFloatComplex t = hipCmulf(w, data[base + k + j + mh]);
     data[base + k + j]      = hipCaddf(u, t);
     data[base + k + j + mh] = hipCsubf(u, t);
 }
 
 /* ---------- column-wise butterfly -------------------------------- */
 __global__ void col_fft_stage_kernel(hipFloatComplex* data,
                                      std::size_t     rows,
                                      std::size_t     cols,
                                      int             stage,
                                      float           dir)
 {
     std::size_t halfPerCol = rows >> 1;
     std::size_t tid  = blockIdx.x * blockDim.x + threadIdx.x;
     std::size_t work = cols * halfPerCol;
     if (tid >= work) return;
 
     std::size_t col = tid / halfPerCol;
     std::size_t off = tid % halfPerCol;
 
     std::size_t mh = 1u << (stage - 1);
     std::size_t m  = mh << 1;
 
     std::size_t k = (off / mh) * m;
     std::size_t j =  off % mh;
 
     float angle = dir * M_PI * static_cast<float>(j) / static_cast<float>(mh);
     float sr, cr;  sincosf(angle, &sr, &cr);
     hipFloatComplex w = make_hipFloatComplex(cr, sr);
 
     std::size_t idxA = (k + j)      * cols + col;
     std::size_t idxB = (k + j + mh) * cols + col;
 
     hipFloatComplex u = data[idxA];
     hipFloatComplex t = hipCmulf(w, data[idxB]);
     data[idxA] = hipCaddf(u, t);
     data[idxB] = hipCsubf(u, t);
 }
 
 /* ---------- bit-reverse reorder kernels (unchanged) -------------- */
 __global__ void bit_reverse_kernel(const hipFloatComplex* in,
                                    hipFloatComplex*       out,
                                    int                   lgN)
 {
     unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned n = 1u << lgN;
     if (i >= n) return;
     out[i] = in[bit_reverse(i, lgN)];
 }
 
 __global__ void row_bit_reverse_kernel(const hipFloatComplex* in,
                                        hipFloatComplex*       out,
                                        std::size_t           rows,
                                        std::size_t           cols,
                                        int                   lgCols)
 {
     std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
     std::size_t total = rows * cols;
     if (idx >= total) return;
 
     std::size_t row = idx / cols;
     std::size_t col = idx % cols;
     unsigned rev = bit_reverse(static_cast<unsigned>(col), lgCols);
     out[row * cols + rev] = in[idx];
 }
 
 __global__ void col_bit_reverse_kernel(const hipFloatComplex* in,
                                        hipFloatComplex*       out,
                                        std::size_t           rows,
                                        std::size_t           cols,
                                        int                   lgRows)
 {
     std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
     std::size_t total = rows * cols;
     if (idx >= total) return;
 
     std::size_t row = idx / cols;
     std::size_t col = idx % cols;
     unsigned rev = bit_reverse(static_cast<unsigned>(row), lgRows);
     out[rev * cols + col] = in[idx];
 }
 
 /* ---------- scaling kernel (used only for inverse) --------------- */
 __global__ void scale_kernel(hipFloatComplex* data,
                              std::size_t     N,
                              float           s)
 {
     unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i >= N) return;
     data[i].x *= s;
     data[i].y *= s;
 }
 
 
 /* ================================================================== *
                      GPU HOST-SIDE  (1-D  forward / inverse)
  * ================================================================== */
 static int fft_gpu_dir(const std::complex<float>* host_in,
                        std::complex<float>*       host_out,
                        std::size_t                N,
                        float                      dir)
 {
     if (N == 0 || (N & (N - 1))) return -1;
 
     int    lgN   = static_cast<int>(std::log2(N));
     size_t bytes = N * sizeof(hipFloatComplex);
 
     hipFloatComplex *d_in{}, *d_data{};
     hipMalloc(&d_in,   bytes);
     hipMalloc(&d_data, bytes);
     hipMemcpy(d_in, host_in, bytes, hipMemcpyHostToDevice);
 
     /* --- bit-reverse copy --------------------------------------- */
     dim3 threads(256), blocks((N + 255) / 256);
     bit_reverse_kernel<<<blocks, threads>>>(d_in, d_data, lgN);
 
     /* --- iterative stages --------------------------------------- */
     size_t half = N >> 1;
     dim3 blocks2((half + 255) / 256);
     for (int s = 1; s <= lgN; ++s)
         fft_stage_kernel<<<blocks2, threads>>>(d_data, N, s, dir);
 
     /* --- scale if inverse --------------------------------------- */
     if (dir > 0.f)
         scale_kernel<<<blocks, threads>>>(d_data, N,
                                           1.f / static_cast<float>(N));
 
     /* --- copy back & cleanup ------------------------------------ */
     hipMemcpy(host_out, d_data, bytes, hipMemcpyDeviceToHost);
     hipFree(d_in);   hipFree(d_data);
     return 0;
 }
 
 int fft_gpu (const std::complex<float>* host_in,
              std::complex<float>*       host_out,
              std::size_t                N)
 { return fft_gpu_dir(host_in, host_out, N, -1.f); }
 
 int ifft_gpu(const std::complex<float>* host_in,
              std::complex<float>*       host_out,
              std::size_t                N)
 { return fft_gpu_dir(host_in, host_out, N, +1.f); }
 
 
 /* ================================================================== *
               GPU HOST-SIDE  (2-D  forward / inverse)                 *
  * ================================================================== */
 static int fft2d_gpu_dir(const std::complex<float>* host_in,
                          std::complex<float>*       host_out,
                          std::size_t                rows,
                          std::size_t                cols,
                          float                      dir)
 {
     if (rows == 0 || cols == 0
         || (rows & (rows - 1)) || (cols & (cols - 1)))
         return -1;
 
     int lgRows = static_cast<int>(std::log2(rows));
     int lgCols = static_cast<int>(std::log2(cols));
 
     std::size_t total = rows * cols;
     std::size_t bytes = total * sizeof(hipFloatComplex);
 
     hipFloatComplex *d_in{}, *d_data1{}, *d_data2{};
     hipMalloc(&d_in,    bytes);
     hipMalloc(&d_data1, bytes);
     hipMalloc(&d_data2, bytes);
     hipMemcpy(d_in, host_in, bytes, hipMemcpyHostToDevice);
 
     dim3 threads(256);
     dim3 blocksTot((total + 255) / 256);
 
     /* --- row bit-reverse --------------------------------------- */
     row_bit_reverse_kernel<<<blocksTot, threads>>>
         (d_in, d_data1, rows, cols, lgCols);
 
     /* --- row stages -------------------------------------------- */
     std::size_t halfRowWork = rows * (cols >> 1);
     dim3 blocksRow((halfRowWork + 255) / 256);
     for (int s = 1; s <= lgCols; ++s)
         row_fft_stage_kernel<<<blocksRow, threads>>>
             (d_data1, rows, cols, s, dir);
 
     /* --- column bit-reverse ------------------------------------ */
     col_bit_reverse_kernel<<<blocksTot, threads>>>
         (d_data1, d_data2, rows, cols, lgRows);
 
     /* --- column stages ----------------------------------------- */
     std::size_t halfColWork = (rows >> 1) * cols;
     dim3 blocksCol((halfColWork + 255) / 256);
     for (int s = 1; s <= lgRows; ++s)
         col_fft_stage_kernel<<<blocksCol, threads>>>
             (d_data2, rows, cols, s, dir);
 
     /* --- scale if inverse -------------------------------------- */
     if (dir > 0.f)
         scale_kernel<<<blocksTot, threads>>>
             (d_data2, total, 1.f / static_cast<float>(total));
 
     hipMemcpy(host_out, d_data2, bytes, hipMemcpyDeviceToHost);
     hipFree(d_in); hipFree(d_data1); hipFree(d_data2);
     return 0;
 }
 
 int fft2d_gpu (const std::complex<float>* host_in,
                std::complex<float>*       host_out,
                std::size_t                rows,
                std::size_t                cols)
 { return fft2d_gpu_dir(host_in, host_out, rows, cols, -1.f); }
 
 int ifft2d_gpu(const std::complex<float>* host_in,
                std::complex<float>*       host_out,
                std::size_t                rows,
                std::size_t                cols)
 { return fft2d_gpu_dir(host_in, host_out, rows, cols, +1.f); }
 
 /* ---------- element-wise (Hadamard) product ----------------------- */
/*  C[i] = A[i] * B[i]   for 0 ≤ i < N                                */
__global__ void pointwise_mul_kernel(const hipFloatComplex* A,
    const hipFloatComplex* B,
    hipFloatComplex*       C,
    std::size_t           N)
{
unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
if (i >= N) return;
C[i] = hipCmulf(A[i], B[i]);
}

/* ---------- element–wise product for 2-D FFT data ------------------- */
/*  C[idx] = A[idx] * B[idx]  for idx = 0 … rows*cols-1                 */
__global__ void pointwise_mul_2d_kernel(const hipFloatComplex* A,
    const hipFloatComplex* B,
    hipFloatComplex*       C,
    std::size_t           total)
{
unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < total)
C[idx] = hipCmulf(A[idx], B[idx]);
}
/*  (You may also just call pointwise_mul_kernel with total=Nrows*Ncols) */
