
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#define CUDA_CHECK(call)                                                     \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      fprintf(stderr, "CUDA error at %s:%d: %s\n",                            \
              __FILE__, __LINE__, hipGetErrorString(err));                   \
      exit(EXIT_FAILURE);                                                     \
    }                                                                         \
  } while (0)

// CUDA kernel function
__global__ void poly_sum_kernel(int *input1, int *input2, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        input2[tid] = input1[tid] + input2[tid];
    }
}

extern "C" int get_max_threads_per_block() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, /*device=*/0);
    return prop.maxThreadsPerBlock;
}

// Wrapper function to call the CUDA kernel
extern "C" void poly_sum(int *input1, int *input2, int size) {
    // Allocate device memory
    int *d_input1, *d_input2;
    CUDA_CHECK(hipMalloc((void**)&d_input1, size * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_input2, size * sizeof(int)));

    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_input1, input1, size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_input2, input2, size * sizeof(int), hipMemcpyHostToDevice));

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Query max active blocks per multiprocessor
    int device;
    CUDA_CHECK(hipGetDevice(&device));

    hipFuncAttributes attr;
    CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(poly_sum_kernel)));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));

    std::cout << prop.maxGridSize[0];
    // In case number of needed blocks exceeds hardware limit
    blocksPerGrid = std::min(blocksPerGrid, prop.maxGridSize[0]);
    poly_sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input1, d_input2, size);

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(input2, d_input2, size * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    hipFree(d_input1);
    hipFree(d_input2);
}
