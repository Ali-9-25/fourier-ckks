
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#define CUDA_CHECK(call)                                                     \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      fprintf(stderr, "CUDA error at %s:%d: %s\n",                            \
              __FILE__, __LINE__, hipGetErrorString(err));                   \
      exit(EXIT_FAILURE);                                                     \
    }                                                                         \
  } while (0)

// CUDA kernel function
__global__ void polynomial_mod_kernel(int *polynomial, int size, int coeff_mod) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        polynomial[tid] %= coeff_mod;
    }
}

// Wrapper function to call the CUDA kernel
extern "C" void polynomial_mod(int *polynomial, int size, int coeff_mod) {
    // Allocate device memory
    int *d_polynomial = nullptr;
    CUDA_CHECK(hipMalloc((void**)&d_polynomial, size * sizeof(int)));

    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_polynomial, polynomial, size * sizeof(int), hipMemcpyHostToDevice));

    // Launch CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Query max active blocks per multiprocessor
    int device;
    CUDA_CHECK(hipGetDevice(&device));

    hipFuncAttributes attr;
    CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(polynomial_mod_kernel)));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));

    std::cout << prop.maxGridSize[0];
    // In case number of needed blocks exceeds hardware limit
    blocksPerGrid = std::min(blocksPerGrid, prop.maxGridSize[0]);
    
    polynomial_mod_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_polynomial, size, coeff_mod);

    CUDA_CHECK(hipDeviceSynchronize());
    //CUDA_CHECK(cudaDeviceSynchronize());
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(polynomial, d_polynomial, size * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(d_polynomial));
}
